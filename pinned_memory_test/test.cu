#include <iostream>
#include <assert.h>

#include <glog/logging.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <sys/mman.h>

#define CUDA_CHECK(condition) \
  /* Code block avoids redefinition of hipError_t error */ \
  do { \
    hipError_t error = condition; \
    CHECK_EQ(error, hipSuccess) << " " << hipGetErrorString(error); \
  } while (0)

#if __CUDA_ARCH__ < 200
    int CUDA_ARCH = 100;
#else
    int CUDA_ARCH = 200;
#endif

using namespace std;

int main() {
  cout << "CUDA_ARCH = " << CUDA_ARCH << endl;

  hipDeviceProp_t deviceProp;
  int devID = 0;
  CUDA_CHECK(hipGetDeviceProperties(&deviceProp, devID));
  cout << "deviceProp.major = " << deviceProp.major << endl;
  cout << "deviceProp.minor = " << deviceProp.minor << endl;
  if (((deviceProp.major << 4) + deviceProp.minor) < 0x20) {
    cout << "binomialOptions requires Compute Capability of SM 2.0 or higher to run.\n";
    hipDeviceReset();
    exit(0);
  }

  size_t count = 1024;
  size_t size = 64 * 1024 * 1024 * sizeof(float);
  // size_t count = 1000;
  // size_t size = 100 * 1000 * 1000 * sizeof(float);
  for (size_t i = 0; i < count; i++) {
    cout << "allocated " << i * 256 << " MB" << endl;
    // cout << i << endl;
    void *host_array;
    // hipHostMalloc(&host_array, size);
    CUDA_CHECK(hipHostMalloc(&host_array, size));
    memset(host_array, 0, size);
    // CUDA_CHECK(hipHostAlloc(&host_array, size, hipHostMallocMapped));
    // CUDA_CHECK(hipMalloc(&host_array, size));
    // CHECK(host_array = malloc(size));
    // CHECK_EQ(mlock(host_array, size), 0);
  }
}