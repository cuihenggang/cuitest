#include <iostream>
#include <assert.h>

#include <hip/hip_runtime.h>


using namespace std;

int main() {
  size_t count = 20 * 1000 * 1000;
  size_t size = count * sizeof(float);
  void *arrays[10];
  for (size_t i = 0; i < 10; i++) {
    assert(hipMalloc(&arrays[i], size) == hipSuccess);
  }
  void *host_array;
  assert(hipHostMalloc(&host_array, size, hipHostMallocDefault) == hipSuccess);
  for (size_t i = 0; i < 10; i++) {
    assert(hipMemset(arrays[8], 0, size) == hipSuccess);
    cout << "memset" << i << endl;
    for (size_t j = 0; j < 10; j++) {
      void *array = arrays[j];
      assert(hipMemcpy(host_array, array, size, hipMemcpyDeviceToHost) == hipSuccess);
      float *floats = reinterpret_cast<float *>(host_array);
      float sum = 0.0;
      for (size_t i = 0; i < count; i++) {
        sum += floats[i];
      }
      cout << "sum" << j << "=" << sum << endl;
    }
  }
}