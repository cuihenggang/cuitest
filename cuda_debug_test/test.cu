#include <iostream>
#include <assert.h>

#include <glog/logging.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CUDA_CHECK(condition) \
  /* Code block avoids redefinition of hipError_t error */ \
  do { \
    hipError_t error = condition; \
    CHECK_EQ(error, hipSuccess) << " " << hipGetErrorString(error); \
  } while (0)

using namespace std;

int main() {
  size_t count = 200 * 1000 * 1000;
  size_t size = count * sizeof(float);
  void *arrays[10];
  for (size_t i = 0; i < 10; i++) {
    CUDA_CHECK(hipMalloc(&arrays[i], size));
  }
  void *host_array;
  CUDA_CHECK(hipHostMalloc(&host_array, size));
  for (size_t i = 0; i < 10; i++) {
    CUDA_CHECK(hipMemset(arrays[8], 0, size));
    cout << "memset" << i << endl;
    for (size_t j = 0; j < 10; j++) {
      void *array = arrays[j];
      CUDA_CHECK(hipMemcpy(host_array, array, size, hipMemcpyDeviceToHost));
      float *floats = reinterpret_cast<float *>(host_array);
      float sum = 0.0;
      for (size_t i = 0; i < count; i++) {
        sum += floats[i];
      }
      cout << "sum" << j << "=" << sum << endl;
    }
  }
  cout << "sum" << "=" << sum << endl;
}