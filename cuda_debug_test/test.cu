#include <iostream>
#include <assert.h>

#include <hip/hip_runtime.h>


using namespace std;

int main() {
  size_t count = 1000 * 20000;
  size_t size = count * sizeof(float);
  void *arrays[10];
  for (size_t i = 0; i < 10; i++) {
    assert(hipMalloc(&arrays[i], size) == hipSuccess);
    assert(hipMemset(arrays[i], 0, size) == hipSuccess);
  }
  void *host_array;
  assert(hipHostMalloc(&host_array, size, hipHostMallocDefault) == hipSuccess);
  for (size_t i = 1; i < 10; i++) {
    void *dst_array = arrays[i];
    void *src_array = arrays[0];
    assert(hipMemcpy(dst_array, src_array, size, hipMemcpyDeviceToDevice) == hipSuccess);
    cout << "copied" << i << endl;
    for (size_t j = 0; j < 10; j++) {
      void *array = arrays[j];
      assert(hipMemcpy(host_array, array, size, hipMemcpyDeviceToHost) == hipSuccess);
      float *floats = reinterpret_cast<float *>(host_array);
      float sum = 0.0;
      for (size_t i = 0; i < count; i++) {
        sum += floats[i];
      }
      cout << "sum" << j << "=" << sum << endl;
    }
  }
}