#include <iostream>
#include <assert.h>

#include <hip/hip_runtime.h>


using namespace std;

int main() {
  size_t count = 200 * 1000 * 1000;
  size_t size = count * sizeof(float);
  void *array;
  assert(hipMalloc(&array, size) == hipSuccess);
  void *host_array;
  assert(hipHostMalloc(&host_array, size, hipHostMallocDefault) == hipSuccess);
  assert(hipMemset(array, 0, size) == hipSuccess);
  assert(hipMemcpy(host_array, array, size, hipMemcpyDeviceToHost) == hipSuccess);
  float *floats = reinterpret_cast<float *>(host_array);
  float sum = 0.0;
  for (size_t i = 0; i < count; i++) {
    sum += floats[i];
  }
  cout << "sum" << "=" << sum << endl;
}